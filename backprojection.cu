#include "hip/hip_runtime.h"
/*
 * Code that uses texture memory to compute a 3D projection of CBCT 
 *
 * IMPORTANT!!! CAUTION!! This code is designed for a Tesla 40k GPU.
 * It is a safe assumption to say that this code wont work in other GPUs as expected
 * or at all. Some of the involved reasons: float/double arithmetic.
 *
 * Ander Biguri
 */

#include <algorithm>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "projection.hpp"
#include "mex.h"
#include <math.h>

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            mexPrintf("%s \n",msg);\
            mexErrMsgIdAndTxt("CBCT:CUDA:interpolation",hipGetErrorString(__err));\
		        } \
	    } while (0)
            

#define MAXTREADS 1024
/*GEOMETRY DEFINITION
 *               
 *                Detector plane, behind
 *            |-----------------------------| 
 *            |                             | 
 *            |                             | 
 *            |                             | 
 *            |                             | 
 *            |      +--------+             |
              |     /        /|             |
     A Z      |    /        / |*D           |
     |        |   +--------+  |             |
     |        |   |        |  |             |
     |        |   |     *O |  +             |
     *--->y   |   |        | /              |
    /         |   |        |/               |
   V X        |   +--------+                |
 *            |-----------------------------|
 *   
 *           *S
 *
 *
 *
 *
 *
 **/
texture<float, hipTextureType3D , hipReadModeElementType> tex; 
    
___global___ void kernelPixelBackprojection(Geometry geo, double* image){
    
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx>= geo.nVoxelX* geo.nVoxelY *geo.nVoxelZ)
        return;
    
    int indVz = idx / (geo.nVoxelX*geo.nVoxelY);
    int resZ  = idx % (geo.nVoxelX*geo.nVoxelY);

    int indY= resZ / geo.nVoxelX;
    int indX= resZ % geo.nVoxelX;
    // Geometric trasnformations:
    
     Point3D S;
     S.x=geo.DSO;
     S.y=0;
     S.z=0;
     
     Point3D P;
     P.x=(indX-(geo.nVoxelX/2-0.5)) *geo.dVoxelX+geo.offOrigX;
     P.y=(indY-(geo.nVoxelY/2-0.5)) *geo.dVoxelY+geo.offOrigY;
     P.z=(indZ-(geo.nVoxelZ/2-0.5)) *geo.dVoxelZ+geo.offOrigZ;
     
     
             
     
    
    
    
    
    
    
    
    
    
    
}
    
    
int projection(float const * const projections, Geometry geo, double*** result,double const * const alphas,int nalpha){
 
    // BEFORE DOING ANYTHING: Use the proper CUDA enabled GPU: Tesla K40c
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
     if (deviceCount == 0)
    {
        mexErrMsgIdAndTxt("CBCT:CUDA:Ax:hipGetDeviceCount","No CUDA enabled NVIDIA GPUs found");
    }
    bool found=false;
    for (int dev = 0; dev < deviceCount; ++dev)
    {
        hipSetDevice(dev);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        if (strcmp(deviceProp.name, "Tesla K40c") == 0){
            found=true;
            break;
        }
    }
    if (!found)
        mexErrMsgIdAndTxt("CBCT:CUDA:Ax:cudaDevice","No Tesla K40c found");

    
    // Done, Tesla found.
    
    /*
     * Allocate texture memory on the device
     */
    
    // copy data to CUDA memory
    hipArray *d_projectiondata = 0;

    const hipExtent extent = make_hipExtent(geo.nDetecU,geo.nDetecV,nalpha);
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	hipMalloc3DArray(&d_projectiondata, &channelDesc, extent);
	cudaCheckErrors("hipMalloc3D error 3D tex");
    
    hipMemcpy3DParms copyParams = { 0 };
	copyParams.srcPtr = make_hipPitchedPtr((void*)projections, extent.width*sizeof(float), extent.width, extent.height);
	copyParams.dstArray = d_projectiondata;
	copyParams.extent = extent;
	copyParams.kind = hipMemcpyHostToDevice;
	hipMemcpy3D(&copyParams);
    
	cudaCheckErrors("hipMemcpy3D fail");
    
    // Configure texture options
    tex.normalized = false;
	tex.filterMode = hipFilterModeLinear;
	tex.addressMode[0] = hipAddressModeBorder;
	tex.addressMode[1] = hipAddressModeBorder;
	tex.addressMode[2] = hipAddressModeBorder;
    
    hipBindTextureToArray(tex, d_projectiondata, channelDesc);    
       
	cudaCheckErrors("3D texture memory bind fail"); 
    
    
    // Allocate result image memory
    size_t num_bytes = geo.nVoxelX*geo.nVoxelY*geo.nVoxelZ * sizeof(double);
    double* dProjection;
    hipMalloc((void**)&dProjection, num_bytes);
    hipMemset(dProjection,0,num_bytes);
    cudaCheckErrors("hipMalloc fail");
    
    
}


