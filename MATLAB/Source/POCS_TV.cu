#include "hip/hip_runtime.h"
/*-------------------------------------------------------------------------
 *
 * CUDA functions for Steepest descend in POCS-type algorithms.
 *
 * This file will iteratively minimize by stepest descend the total variation
 * of the input image, with the parameters given, using GPUs.
 *
 * CODE by       Ander Biguri
 *
 * ---------------------------------------------------------------------------
 * ---------------------------------------------------------------------------
 * Copyright (c) 2015, University of Bath and CERN- European Organization for
 * Nuclear Research
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice,
 * this list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its contributors
 * may be used to endorse or promote products derived from this software without
 * specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 * ---------------------------------------------------------------------------
 *
 * Contact: tigre.toolbox@gmail.com
 * Codes  : https://github.com/CERN/TIGRE
 * ---------------------------------------------------------------------------
 */







#define MAXTHREADS 1024

#include "POCS_TV.hpp"




#define cudaCheckErrors(msg) \
do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
                mexPrintf("%s \n",msg);\
                hipDeviceReset();\
                        mexErrMsgIdAndTxt("CBCT:CUDA:POCS_TV",hipGetErrorString(__err));\
        } \
} while (0)
    
// CUDA kernels
//https://stackoverflow.com/questions/21332040/simple-cuda-kernel-optimization/21340927#21340927
    __global__ void divideArrayScalar(float* vec,float scalar,const size_t n){
        unsigned long long i = (blockIdx.x * blockDim.x) + threadIdx.x;
        for(; i<n; i+=gridDim.x*blockDim.x) {
            vec[i]/=scalar;
        }
    }
    __global__ void multiplyArrayScalar(float* vec,float scalar,const size_t n)
    {
        unsigned long long i = (blockIdx.x * blockDim.x) + threadIdx.x;
        for(; i<n; i+=gridDim.x*blockDim.x) {
            vec[i]*=scalar;
        }
    }
    __global__ void substractArrays(float* vec,float* vec2,const size_t n)
    {
        unsigned long long i = (blockIdx.x * blockDim.x) + threadIdx.x;
        for(; i<n; i+=gridDim.x*blockDim.x) {
            vec[i]-=vec2[i];
        }
    }
    
    __device__ __inline__
            void gradient(const float* u, float* grad,
            long z, long y, long x,
            long depth, long rows, long cols){
        unsigned long size2d = rows*cols;
        unsigned long long idx = z * size2d + y * cols + x;
        
        float uidx = u[idx];
        
        if ( z - 1 >= 0 && z<depth) {
            grad[0] = (uidx-u[(z-1)*size2d + y*cols + x]) ;
        }
        
        if ( y - 1 >= 0 && y<rows){
            grad[1] = (uidx-u[z*size2d + (y-1)*cols + x]) ;
        }
        
        if ( x - 1 >= 0 && x<cols) {
            grad[2] = (uidx-u[z*size2d + y*cols + (x-1)]);
        }
    }
    
    __global__ void gradientTV(const float* f, float* dftv,
            long depth, long rows, long cols){
        unsigned long x = threadIdx.x + blockIdx.x * blockDim.x;
        unsigned long y = threadIdx.y + blockIdx.y * blockDim.y;
        unsigned long z = threadIdx.z + blockIdx.z * blockDim.z;
        unsigned long long idx = z * rows * cols + y * cols + x;
        if ( x >= cols || y >= rows || z >= depth )
            return;
        
        
        float df[3] ={0.f,0.f,0.f};
        float dfi[3]={0.f,0.f,0.f}; // dfi== \partial f_{i+1,j,k}
        float dfj[3]={0.f,0.f,0.f};
        float dfk[3]={0.f,0.f,0.f};
        gradient(f,df  ,z  ,y  ,x  , depth,rows,cols);
        gradient(f,dfi ,z  ,y  ,x+1, depth,rows,cols);
        gradient(f,dfj ,z  ,y+1,x  , depth,rows,cols);
        gradient(f,dfk ,z+1,y  ,x  , depth,rows,cols);
        float eps=0.00000001; //% avoid division by zero
        
        dftv[idx]=(df[0]+df[1]+df[2])/(sqrt(df[0] *df[0] +df[1] *df[1] +df[2] *df[2])+eps)
        -dfi[2]/(sqrt(dfi[0]*dfi[0]+dfi[1]*dfi[1]+dfi[2]*dfi[2]) +eps)     // I wish I coudl precompute this, but if I do then Id need to recompute the gradient.
        -dfj[1]/(sqrt(dfj[0]*dfj[0]+dfj[1]*dfj[1]+dfj[2]*dfj[2]) +eps)
        -dfk[0]/(sqrt(dfk[0]*dfk[0]+dfk[1]*dfk[1]+dfk[2]*dfk[2]) +eps);
        return;
        
    }
    
    __device__ void warpReduce(volatile float *sdata, size_t tid) {
        sdata[tid] += sdata[tid + 32];
        sdata[tid] += sdata[tid + 16];
        sdata[tid] += sdata[tid + 8];
        sdata[tid] += sdata[tid + 4];
        sdata[tid] += sdata[tid + 2];
        sdata[tid] += sdata[tid + 1];
    }
    
    __global__ void  reduceNorm2(float *g_idata, float *g_odata, size_t n){
        extern __shared__ volatile float sdata[];
        //http://stackoverflow.com/a/35133396/1485872
        size_t tid = threadIdx.x;
        size_t i = blockIdx.x*blockDim.x + tid;
        size_t gridSize = blockDim.x*gridDim.x;
        float mySum = 0;
        float value=0;
        while (i < n) {
            value=g_idata[i]; //avoid reading twice
            mySum += value*value;
            i += gridSize;
        }
        sdata[tid] = mySum;
        __syncthreads();
        
        if (tid < 512)
            sdata[tid] += sdata[tid + 512];
        __syncthreads();
        if (tid < 256)
            sdata[tid] += sdata[tid + 256];
        __syncthreads();
        
        if (tid < 128)
            sdata[tid] += sdata[tid + 128];
        __syncthreads();
        
        if (tid <  64)
            sdata[tid] += sdata[tid + 64];
        __syncthreads();
        
        
#if (__CUDA_ARCH__ >= 300)
        if ( tid < 32 )
        {
            mySum = sdata[tid] + sdata[tid + 32];
            for (int offset = warpSize/2; offset > 0; offset /= 2) {
                mySum += __shfl_down(mySum, offset);
            }
        }
#else
        if (tid < 32) {
            warpReduce(sdata, tid);
            mySum = sdata[0];
        }
#endif
        if (tid == 0) g_odata[blockIdx.x] = mySum;
    }
    
    __global__ void  reduceSum(float *g_idata, float *g_odata, size_t n){
        extern __shared__ volatile float sdata[];
        //http://stackoverflow.com/a/35133396/1485872
        size_t tid = threadIdx.x;
        size_t i = blockIdx.x*blockDim.x + tid;
        size_t gridSize = blockDim.x*gridDim.x;
        float mySum = 0;
        // float value=0;
        while (i < n) {
            mySum += g_idata[i];
            i += gridSize;
        }
        sdata[tid] = mySum;
        __syncthreads();
        
        if (tid < 512)
            sdata[tid] += sdata[tid + 512];
        __syncthreads();
        if (tid < 256)
            sdata[tid] += sdata[tid + 256];
        __syncthreads();
        
        if (tid < 128)
            sdata[tid] += sdata[tid + 128];
        __syncthreads();
        
        if (tid <  64)
            sdata[tid] += sdata[tid + 64];
        __syncthreads();
        
        
#if (__CUDA_ARCH__ >= 300)
        if ( tid < 32 )
        {
            mySum = sdata[tid] + sdata[tid + 32];
            for (int offset = warpSize/2; offset > 0; offset /= 2) {
                mySum += __shfl_down(mySum, offset);
            }
        }
#else
        if (tid < 32) {
            warpReduce(sdata, tid);
            mySum = sdata[0];
        }
#endif
        if (tid == 0) g_odata[blockIdx.x] = mySum;
    }
    
    
    
    
// main function
    void pocs_tv(const float* img,float* dst,float alpha,const long* image_size, int maxIter){
        
        
        
        
        // Prepare for MultiGPU
        int deviceCount = 0;
        hipGetDeviceCount(&deviceCount);
        cudaCheckErrors("Device query fail");
        if (deviceCount == 0) {
            mexErrMsgIdAndTxt("minimizeTV:POCS_TV:GPUselect","There are no available device(s) that support CUDA\n");
        }
        //
        // CODE assumes
        // 1.-All available devices are usable by this code
        // 2.-All available devices are equal, they are the same machine (warning trhown)
        int dev;
        char * devicenames;
        hipDeviceProp_t deviceProp;
        
        for (dev = 0; dev < deviceCount; dev++) {
            hipSetDevice(dev);
            hipGetDeviceProperties(&deviceProp, dev);
            if (dev>0){
                if (strcmp(devicenames,deviceProp.name)!=0){
                    mexWarnMsgIdAndTxt("minimizeTV:POCS_TV:GPUselect","Detected one (or more) different GPUs.\n This code is not smart enough to separate the memory GPU wise if they have different computational times or memory limits.\n First GPU parameters used. If the code errors you might need to change the way GPU selection is performed. \n POCS_TV.cu line 277.");
                    break;
                }
            }
            devicenames=deviceProp.name;
        }
        
        
        // We don't know if the devices are being used. lets check that. and only use the amount of memory we need.
        size_t memfree;
        size_t memtotal;
        size_t mem_GPU_global;
        for (dev = 0; dev < deviceCount; dev++){
            hipSetDevice(dev);
            hipMemGetInfo(&memfree,&memtotal);
            if(dev==0) mem_GPU_global=memfree;
            if(memfree<memtotal/2){
                mexErrMsgIdAndTxt("minimizeTV:POCS_TV:GPU","One (or more) of your GPUs is being heavily used by another program (possibly graphics-based).\n Free the GPU to run TIGRE\n");
            }
            cudaCheckErrors("Check mem error");
            
            mem_GPU_global=(memfree<mem_GPU_global)?memfree:mem_GPU_global;
        }
        mem_GPU_global=(size_t)((double)mem_GPU_global*0.95);
        
        
        // %5 of free memory shoudl be enough, we have almsot no variables in these kernels
        size_t total_pixels              = image_size[0] * image_size[1]  * image_size[2] ;
        size_t mem_slice_image           = sizeof(float)* image_size[0] * image_size[1]  ;
        size_t mem_size_image            = sizeof(float)* total_pixels;
        size_t mem_auxiliary             = sizeof(float)* (total_pixels + MAXTHREADS - 1) / MAXTHREADS;
        
        // Decide how are we handling the distribution of computation
        size_t mem_img_each_GPU;
        
        unsigned int buffer_length=2;
        //Does everything fit in the GPU?
        bool fits_in_memory=false;
        unsigned int slices_per_split;
        unsigned int splits=1; // if the number does not fit in an uint, you have more serious trouble than this.
        if(mem_GPU_global> 3*mem_size_image+3*(deviceCount-1)*mem_slice_image+mem_auxiliary){
            // We only need to split if we have extra GPUs
            fits_in_memory=true;
            slices_per_split=(image_size[2]+deviceCount-1)/deviceCount;
            mem_img_each_GPU=mem_slice_image*((image_size[2]+buffer_length*2+deviceCount-1)/deviceCount);
        }else{
            fits_in_memory=false;
            // As mem_auxiliary is not expected to be a large value (for a 2000^3 image is around 28Mbytes), lets for now assume we need it all
            size_t mem_free=mem_GPU_global-mem_auxiliary;
            
            splits=(unsigned int)(ceil(((float)(3*mem_size_image)/(float)(deviceCount))/mem_free));
            // Now, there is an overhead here, as each splits should have 2 slices more, to accoutn for overlap of images.
            // lets make sure these 2 slices fit, if they do not, add 1 to splits.
            slices_per_split=(image_size[2]+deviceCount*splits-1)/(deviceCount*splits);
            mem_img_each_GPU=(mem_slice_image*(slices_per_split+buffer_length*2));
            
            // if the new stuff does not fit in the GPU, it measn we are in the edge case where adding that extra slice will overflow memory
            if (mem_GPU_global< 3*mem_img_each_GPU+mem_auxiliary){
                // one more splot shoudl do the job, as its an edge case.
                splits++;
                //recompute for later
                slices_per_split=(image_size[2]+deviceCount*splits-1)/(deviceCount*splits); // amountf of slices that fit on a GPU. Later we add 2 to these, as we need them for overlap
                mem_img_each_GPU=(mem_slice_image*(slices_per_split+buffer_length*2));
            }

            
            
            // How many EXTRA buffer slices shoudl be able to fit in here??!?!
            mem_free=mem_GPU_global-(3*mem_img_each_GPU+mem_auxiliary);
            unsigned int extra_buff=(mem_free/mem_slice_image); 
            buffer_length=(extra_buff/2)/3; // we need double whatever this results in, rounded down.
            mem_img_each_GPU=(mem_slice_image*(slices_per_split+buffer_length*2));

            // Assert
            if (mem_GPU_global< 3*mem_img_each_GPU+mem_auxiliary){
                mexErrMsgIdAndTxt("minimizeTV:POCS_TV:GPU","Bad assert. Logic behind spliting flawed! Please tell: ander.biguri@gmail.com\n");
            }
        }
        
        
//         mexPrintf("mem_img_each_GPU :%zu\n",mem_img_each_GPU);
//         mexPrintf("mem_GPU_global :%zu\n",mem_GPU_global);
//         mexPrintf("mem_auxiliary :%zu\n",mem_auxiliary);
//         mexPrintf("slices_per_split*mem_slice_image :%zu\n",(size_t)(slices_per_split*mem_slice_image));
//         mexPrintf("splits :%u\n",splits);
//         mexPrintf("deviceCount :%u\n",deviceCount);
        
//         return;
        float** d_image=    (float**)malloc(deviceCount*sizeof(float*));
        float** d_dimgTV=   (float**)malloc(deviceCount*sizeof(float*));
        float** d_norm2aux= (float**)malloc(deviceCount*sizeof(float*));
        float** d_norm2=    (float**)malloc(deviceCount*sizeof(float*));
        
        // allocate memory in each GPU
        for (dev = 0; dev < deviceCount; dev++){
            hipSetDevice(dev);
            
            
            hipMalloc((void**)&d_image[dev]    , mem_img_each_GPU);
            hipMemset(d_image[dev],0,mem_img_each_GPU);
            hipMalloc((void**)&d_dimgTV[dev]   , mem_img_each_GPU);
            hipMalloc((void**)&d_norm2[dev]    , slices_per_split*mem_slice_image);
            hipMalloc((void**)&d_norm2aux[dev] , mem_auxiliary);
            cudaCheckErrors("Malloc  error");
            
            
        }
        
        float* buffer;
        if(splits>1){
            mexWarnMsgIdAndTxt("minimizeTV:POCS_TV:Image_split","Your image can not be fully split between the available GPUs. The computation of minTV will be significantly slowed due to the image size.\nApproximated mathematics turned on for computational speed.");
        }else{
            buffer=(float*)malloc(image_size[0]*image_size[1]*2*sizeof(float));
        }
        
        
        
        // For the reduction
        
        
        double totalsum_prev;
        double totalsum;
        float sum_curr_spl;
        
        float* sumnorm2=(float*)malloc(deviceCount*sizeof(float));
        
        unsigned int curr_slices;
        unsigned long long curr_pixels;
        unsigned long long buffer_pixels=buffer_length*image_size[0]*image_size[1];
        
        for(unsigned int i=0;i<maxIter;i+=(buffer_length-1)){
            if(splits>1){
                totalsum_prev=0;
            }
            for(unsigned int sp=0;sp<splits;sp++){
                
                // For each iteration we need to comptue all the image. The ordering of these loops
                // need to be like this due to the boudnign layers between slpits. If more than 1 split is needed
                // for each GPU then there is no other way that taking the entire memory out of GPU and putting it back.
                // If the memory can be shared ebtween GPUs fully without extra splits, then there is an easy way of syncronizing the memory
                
                // Copy image to memory
                size_t linear_idx_start;
                if(i==0){
                    for (dev = 0; dev < deviceCount; dev++){
                        curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                        curr_pixels=curr_slices*image_size[0]*image_size[1];
                        linear_idx_start=image_size[0]*image_size[1]*slices_per_split*(sp*deviceCount+dev);
                        
                        hipSetDevice(dev);
                        hipMemcpyAsync(d_image[dev]+buffer_pixels, &img[linear_idx_start], curr_pixels*sizeof(float), hipMemcpyHostToDevice);
                        // if its not the last, copy also the intersection buffer.
                        if((sp*deviceCount+dev)<deviceCount*splits-1){
                            hipMemcpyAsync(d_image[dev]+curr_pixels+buffer_pixels, &img[linear_idx_start]+curr_pixels, buffer_pixels*sizeof(float), hipMemcpyHostToDevice);
                        }
                        // if its not the first, copy also the intersection buffer.
                        if((sp*deviceCount+dev)){
                            hipMemcpyAsync(d_image[dev], &img[linear_idx_start]-buffer_pixels, buffer_pixels*sizeof(float), hipMemcpyHostToDevice);
                        }
                        
                    }
                }
                hipDeviceSynchronize();
                cudaCheckErrors("Memcpy failure");
                
                // if we need to split and its not the first iteration, then we need to copy from Host memory the previosu result.
                if (splits>1 & i>0){
                    for (dev = 0; dev < deviceCount; dev++){
                        curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                        linear_idx_start=image_size[0]*image_size[1]*slices_per_split*(sp*deviceCount+dev);
                        curr_pixels=curr_slices*image_size[0]*image_size[1];
                        
                        hipSetDevice(dev);
                        hipMemcpyAsync(d_image[dev]+buffer_pixels, &dst[linear_idx_start], curr_pixels*sizeof(float), hipMemcpyHostToDevice);
                        
                        // if its not the last, copy also the intersection buffer.
                        if((sp*deviceCount+dev)<deviceCount*splits-1){
                            
                            hipMemcpyAsync(d_image[dev]+curr_pixels+buffer_pixels, &dst[linear_idx_start]+curr_pixels, buffer_pixels*sizeof(float), hipMemcpyHostToDevice);
                        }
                        // if its not the first, copy also the intersection buffer.
                        if((sp*deviceCount+dev)){
                            hipMemcpyAsync(d_image[dev], &dst[linear_idx_start]-buffer_pixels,buffer_pixels*sizeof(float), hipMemcpyHostToDevice);
                        }
                        
                    }
                    
                }
                hipDeviceSynchronize();
                cudaCheckErrors("Memcpy failure on multi split");
                for(unsigned int ib=0;  (ib<(buffer_length-1)) && ((i+ib)<maxIter);  ib++){
                    
                    // For the gradient
                    dim3 blockGrad(10, 10, 10);
                    dim3 gridGrad((image_size[0]+blockGrad.x-1)/blockGrad.x, (image_size[1]+blockGrad.y-1)/blockGrad.y, (curr_slices+buffer_length*2+blockGrad.z-1)/blockGrad.z);
                    
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(dev);
                        curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                        // Compute the gradient of the TV norm
                        
                        // I Dont understand why I need to store 2 layers to compute correctly with 1 buffer. The bounding checks shoudl
                        // be enough but they are not.
                        gradientTV<<<gridGrad, blockGrad>>>(d_image[dev],d_dimgTV[dev],(long)(curr_slices+buffer_length*2-1), image_size[1],image_size[0]);
                        
                    }
                    
                    hipDeviceSynchronize();
                    cudaCheckErrors("Gradient");
                    
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(dev);
                        curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                        // no need to copy the 2 aux slices here
                        hipMemcpyAsync(d_norm2[dev], d_dimgTV[dev]+buffer_pixels, image_size[0]*image_size[1]*curr_slices*sizeof(float), hipMemcpyDeviceToDevice);
                    }
                    hipDeviceSynchronize();
                    cudaCheckErrors("Copy from gradient call error");
                    
                    
                    // Compute the L2 norm of the gradint. For that, reduction is used.
                    //REDUCE
                    float test;
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(dev);
                        curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                        
                        total_pixels=curr_slices*image_size[0]*image_size[1];
                        
                        size_t dimblockRed = MAXTHREADS;
                        size_t dimgridRed = (total_pixels + MAXTHREADS - 1) / MAXTHREADS;
                        reduceNorm2 << <dimgridRed, dimblockRed, MAXTHREADS*sizeof(float) >> >(d_norm2[dev], d_norm2aux[dev], total_pixels);
                        //hipDeviceSynchronize();
                        //cudaCheckErrors("reduce1");
                        if (dimgridRed > 1) {
                            reduceSum << <1, dimblockRed, MAXTHREADS*sizeof(float) >> >(d_norm2aux[dev], d_norm2[dev], dimgridRed);
                            //cudaCheckErrors("reduce2");
                            //hipDeviceSynchronize();
                            hipMemcpyAsync(&sumnorm2[dev], d_norm2[dev], sizeof(float), hipMemcpyDeviceToHost);
                            //cudaCheckErrors("hipMemcpy reduce1");
                        }
                        else {
                            hipMemcpyAsync(&sumnorm2[dev], d_norm2aux[dev], sizeof(float), hipMemcpyDeviceToHost);
                            //cudaCheckErrors("hipMemcpy reduce2");
                        }
                    }
                    hipDeviceSynchronize();
                    cudaCheckErrors("Reduction error");
                    
                    
                    
                    sum_curr_spl=0;
                    // this is CPU code
                    for (dev = 0; dev < deviceCount; dev++){
                        sum_curr_spl+=sumnorm2[dev];
                    }
                    sum_curr_spl+=0.0000001f;
                    
                    if(i>0 && splits>1){
                        //totalsum=totalsum_prev;
                    }else{
                        totalsum=sum_curr_spl;
                    }
                    
                    
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(dev);
                        curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                        total_pixels=curr_slices*image_size[0]*image_size[1];
                        //NOMRALIZE
                        //in a Tesla, maximum blocks =15 SM * 4 blocks/SM
                        divideArrayScalar  <<<60,MAXTHREADS>>>(d_dimgTV[dev]+buffer_pixels,(float)sqrt(totalsum),total_pixels);
                        //hipDeviceSynchronize();
                        //cudaCheckErrors("Division error");
                        //MULTIPLY HYPERPARAMETER
                        multiplyArrayScalar<<<60,MAXTHREADS>>>(d_dimgTV[dev]+buffer_pixels,alpha,   total_pixels);
                    }
                    hipDeviceSynchronize();
                    cudaCheckErrors("Scalar operations error");
                    
                    //SUBSTRACT GRADIENT
                    //////////////////////////////////////////////
                    for (dev = 0; dev < deviceCount; dev++){
                        hipSetDevice(dev);
                        curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                        total_pixels=curr_slices*image_size[0]*image_size[1];
                        
                        substractArrays<<<60,MAXTHREADS>>>(d_image[dev]+buffer_pixels,d_dimgTV[dev]+buffer_pixels, total_pixels);
                    }
                    
                }
                // Syncronize mathematics, make sure bounding pixels are correct
                hipDeviceSynchronize();
                if(splits==1){
                    for(dev=0; dev<deviceCount;dev++){
                        if (dev<deviceCount-1){
                            hipSetDevice(dev+1);
                            hipMemcpy(buffer, d_image[dev+1], buffer_pixels*sizeof(float), hipMemcpyDeviceToHost);
                            hipSetDevice(dev);
                            hipMemcpy(d_image[dev]+slices_per_split+buffer_pixels,buffer, buffer_pixels*sizeof(float), hipMemcpyHostToDevice);
                            
                            
                        }
                        if (dev>0){
                            hipSetDevice(dev-1);
                            hipMemcpy(buffer, d_image[dev-1]+slices_per_split+buffer_pixels, buffer_pixels*sizeof(float), hipMemcpyDeviceToHost);
                            hipSetDevice(dev);
                            hipMemcpy(d_image[dev],buffer, buffer_pixels*sizeof(float), hipMemcpyHostToDevice);
                        }
                    }
                }else{
                    // We need to take it out :(
                    for(dev=0; dev<deviceCount;dev++){
                        hipSetDevice(dev);
                        
                        curr_slices=((sp*deviceCount+dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*(sp*deviceCount+dev);
                        linear_idx_start=image_size[0]*image_size[1]*slices_per_split*(sp*deviceCount+dev);
                        total_pixels=curr_slices*image_size[0]*image_size[1];
//                         mexPrintf("curr_slices %u \n",curr_slices);
//                         mexPrintf("linear_idx_start %llu \n",linear_idx_start);
//                         mexPrintf("total_pixels %llu \n",total_pixels);
                        
                        hipMemcpy(&dst[linear_idx_start], d_image[dev]+buffer_pixels,total_pixels*sizeof(float), hipMemcpyDeviceToHost);
                    }
                }
                hipDeviceSynchronize();
                cudaCheckErrors("Memory gather error");
                
                totalsum_prev+=sum_curr_spl;
            }
            totalsum=totalsum_prev;
        }
        // If there has not been splits, we still have data in memory
        if(splits==1){
            for(dev=0; dev<deviceCount;dev++){
                hipSetDevice(dev);
                
                curr_slices=((dev+1)*slices_per_split<image_size[2])?  slices_per_split:  image_size[2]-slices_per_split*dev;
                total_pixels=curr_slices*image_size[0]*image_size[1];
                hipMemcpy(dst+slices_per_split*dev, d_image[dev]+buffer_pixels,total_pixels*sizeof(float), hipMemcpyDeviceToHost);
            }
        }
        cudaCheckErrors("Copy result back");
        
        for(dev=0; dev<deviceCount;dev++){
            hipSetDevice(dev);
            hipFree(d_image[dev]);
            hipFree(d_norm2aux[dev]);
            hipFree(d_dimgTV[dev]);
            hipFree(d_norm2[dev]);
        }
        cudaCheckErrors("Memory free");
        hipDeviceReset();
    }
    
