#include "hip/hip_runtime.h"
/*-------------------------------------------------------------------------
 *
 * CUDA functions for texture-memory interpolation based projection
 *
 * This file has the necesary fucntiosn to perform X-ray CBCT projection
 * operation given a geaometry, angles and image. It uses the 3D texture
 * memory linear interpolation to uniformily sample a path to integrate the
 * X-rays.
 *
 * CODE by       Ander Biguri
 *               Sepideh Hatamikia (arbitrary rotation)
 * ---------------------------------------------------------------------------
 * ---------------------------------------------------------------------------
 * Copyright (c) 2015, University of Bath and CERN- European Organization for
 * Nuclear Research
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice,
 * this list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its contributors
 * may be used to endorse or promote products derived from this software without
 * specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 * ---------------------------------------------------------------------------
 *
 * Contact: tigre.toolbox@gmail.com
 * Codes  : https://github.com/CERN/TIGRE
 * ---------------------------------------------------------------------------
 */






#include <algorithm>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "ray_interpolated_projection.hpp"
#include "mex.h"
#include <math.h>

#define cudaCheckErrors(msg) \
do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
                mexPrintf("%s \n",msg);\
                hipDeviceReset();\
                mexErrMsgIdAndTxt("CBCT:CUDA:Atb",hipGetErrorString(__err));\
        } \
} while (0)
    
    
// Declare the texture reference.

#define MAXTREADS 1024
/*GEOMETRY DEFINITION
 *
 *                Detector plane, behind
 *            |-----------------------------|
 *            |                             |
 *            |                             |
 *            |                             |
 *            |                             |
 *            |      +--------+             |
 *            |     /        /|             |
 *   A Z      |    /        / |*D           |
 *   |        |   +--------+  |             |
 *   |        |   |        |  |             |
 *   |        |   |     *O |  +             |
 *    --->y   |   |        | /              |
 *  /         |   |        |/               |
 * V X        |   +--------+                |
 *            |-----------------------------|
 *
 *           *S
 *
 *
 *
 *
 *
 **/
void CreateTextureInterp(int num_devices,const float* imagedata,Geometry geo,hipArray** d_cuArrTex, hipTextureObject_t *texImage);


__global__ void vecAddInPlaceInterp(float *a, float *b, unsigned long  n)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    // Make sure we do not go out of bounds
    if (idx < n)
        a[idx] = a[idx] + b[idx];
}
    
    
template<bool sphericalrotation>
        __global__ void kernelPixelDetector( Geometry geo,
        float* detector,
        Point3D source ,
        Point3D deltaU,
        Point3D deltaV,
        Point3D uvOrigin,
        float DSO,
        float cropdist_init,
        hipTextureObject_t tex){
    
    unsigned long  y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned long  x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long  idx =  x  * geo.nDetecV + y;
    
    if ((x>= geo.nDetecU) | (y>= geo.nDetecV))
        return;
    
    
    
    
    /////// Get coordinates XYZ of pixel UV
    int pixelV = geo.nDetecV-y-1;
    int pixelU = x;
    
    
    
    float vectX,vectY,vectZ;
    Point3D P;
    P.x=(uvOrigin.x+pixelU*deltaU.x+pixelV*deltaV.x);
    P.y=(uvOrigin.y+pixelU*deltaU.y+pixelV*deltaV.y);
    P.z=(uvOrigin.z+pixelU*deltaU.z+pixelV*deltaV.z);
    
    // Length is the ray length in normalized space
    float length=sqrtf((source.x-P.x)*(source.x-P.x)+(source.y-P.y)*(source.y-P.y)+(source.z-P.z)*(source.z-P.z));
    //now legth is an integer of Nsamples that are required on this line
    length=ceilf(length/geo.accuracy);//Divide the directional vector by an integer
    vectX=(P.x -source.x)/(length);
    vectY=(P.y -source.y)/(length);
    vectZ=(P.z -source.z)/(length);
    
    
//     //Integrate over the line
    float tx,ty,tz;
    float sum=0;
    float i;
    
    

//  Because I have no idea how to efficiently cutoff the legth path in 3D, a very upper limit is computed (see maxdistanceCuboid)
//  for the 3D case. However it would be bad to lose performance in the 3D case
//  TODO: can ge really improve this?
    if (sphericalrotation){
        if ((2*DSO/fminf(fminf(geo.dVoxelX,geo.dVoxelY),geo.dVoxelZ)+cropdist_init)/geo.accuracy  <   length)
            length=ceilf((2*DSO/fminf(fminf(geo.dVoxelX,geo.dVoxelY),geo.dVoxelZ)+cropdist_init)/geo.accuracy);
    }
    else{
        if ((2*DSO/fminf(geo.dVoxelX,geo.dVoxelY)+cropdist_init)/geo.accuracy  <   length)
            length=ceilf((2*DSO/fminf(geo.dVoxelX,geo.dVoxelY)+cropdist_init)/geo.accuracy);
    }

    
    //Length is not actually a length, but the amount of memreads with given accuracy ("samples per voxel")
    for (i=floorf(cropdist_init/geo.accuracy); i<=length; i=i+1){
        tx=vectX*i+source.x;
        ty=vectY*i+source.y;
        tz=vectZ*i+source.z;
        
        sum += tex3D<float>(tex, tx+0.5f, ty+0.5f, tz+0.5f); // this line is 94% of time.
    }
    
    float deltalength=sqrtf((vectX*geo.dVoxelX)*(vectX*geo.dVoxelX)+
                            (vectY*geo.dVoxelY)*(vectY*geo.dVoxelY)+
                            (vectZ*geo.dVoxelZ)*(vectZ*geo.dVoxelZ) );
    
    detector[idx]=sum*deltalength;
}



// legnth(angles)=3 x nagnles, as we have roll, pitch, yaw.
int interpolation_projection(float const * const img, Geometry geo, float** result,float const * const angles,int nangles){
    
    
        
    
    
    
    
    
    
    // Prepare for MultiGPU
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    cudaCheckErrors("Device query fail");
    if (deviceCount == 0) {
        mexErrMsgIdAndTxt("Ax:Interpolated_projection:GPUselect","There are no available device(s) that support CUDA\n");
    }
    //
    // CODE assumes
    // 1.-All available devices are usable by this code
    // 2.-All available devices are equal, they are the same machine (warning trhown)
    int dev;
    char * devicenames;
    hipDeviceProp_t deviceProp;
    
    for (dev = 0; dev < deviceCount; dev++) {
        hipSetDevice(dev);
        hipGetDeviceProperties(&deviceProp, dev);
        if (dev>0){
            if (strcmp(devicenames,deviceProp.name)!=0){
                mexWarnMsgIdAndTxt("Ax:GPUselect","Detected one (or more) different GPUs.\n This code is not smart enough to separate the memory GPU wise if they have different computational times or memory limits.\n First GPU parameters used. If the code errors you might need to change the way GPU selection is performed. \n Siddon_projection.cu line 275.");
                break;
            }
        }
        devicenames=deviceProp.name;
    }
    hipSetDevice(0);
    hipGetDeviceProperties(&deviceProp, 0);
    unsigned long long mem_GPU_global=(unsigned long long)(deviceProp.totalGlobalMem*0.9);
    size_t mem_image=(unsigned long long)geo.nVoxelX*(unsigned long long)geo.nVoxelY*(unsigned long long)geo.nVoxelZ*sizeof(float);
    size_t mem_proj =(unsigned long long)geo.nDetecU*(unsigned long long)geo.nDetecV * sizeof(float);
    
    // Does everything fit in the GPUs?
    bool fits_in_memory=false;
    unsigned int splits=1;
    Geometry * geoArray;
    
  
    if (mem_image+mem_proj<mem_GPU_global){// yes it does
        fits_in_memory=true;
        geoArray=(Geometry*)malloc(sizeof(Geometry));
        geoArray[0]=geo;
    }
    else{// Nope nope.
        fits_in_memory=false; // Oh dear.
        // approx free memory we have. We already have left some extra 10% free for internal stuff
        // we need a second projection memory to combine multi-GPU stuff.
        size_t mem_free=mem_GPU_global-2*mem_proj;
        

        splits=mem_image/mem_free+1;// Ceil of the truncation
        geoArray=(Geometry*)malloc(splits*sizeof(Geometry));
        splitImageInterp(splits,geo,geoArray,nangles);
    }
    
    float ** dProjection_accum;
    size_t num_bytes_proj = geo.nDetecU*geo.nDetecV * sizeof(float);
    if (!fits_in_memory){
        dProjection_accum=(float**)malloc(deviceCount*sizeof(float*));
        for (dev = 0; dev < deviceCount; dev++) {
            hipSetDevice(dev);
            hipMalloc((void**)&dProjection_accum[dev], num_bytes_proj);
            hipMemset(dProjection_accum[dev],0,num_bytes_proj);
            cudaCheckErrors("cudaMallocauxiliarty projections fail");
        }
    }
    
    // This is happening regarthless if the image fits on memory
    float** dProjection=(float**)malloc(deviceCount*sizeof(float*));
    for (dev = 0; dev < deviceCount; dev++) {
        hipSetDevice(dev);
        hipMalloc((void**)&dProjection[dev], num_bytes_proj);
        hipMemset(dProjection[dev],0,num_bytes_proj);
        cudaCheckErrors("hipMalloc projections fail");
    }
    
    
    float cropdist_init;
    for (unsigned int sp=0;sp<splits;sp++){
        
        // Create texture objects for all GPUs
        hipTextureObject_t *texImg = new hipTextureObject_t[deviceCount];
        hipArray **d_cuArrTex = new hipArray*[deviceCount];
        
        size_t linear_idx_start;
        //First one shoudl always be  the same size as all the rest but the last
        linear_idx_start= sp*geoArray[0].nVoxelX*geoArray[0].nVoxelY*geoArray[0].nVoxelZ;
        CreateTextureInterp(deviceCount,&img[linear_idx_start],geoArray[sp],d_cuArrTex,texImg);
        cudaCheckErrors("Texture object creation fail");
        
        
        
        
        Point3D source, deltaU, deltaV, uvOrigin;
        
        
        int divU,divV;
        divU=8;
        divV=8;
        dim3 grid((geoArray[sp].nDetecU+divU-1)/divU,(geoArray[0].nDetecV+divV-1)/divV,1);
        dim3 block(divU,divV,1);
        
        
        
        
        for (unsigned int i=0;i<nangles;i+=(unsigned int)deviceCount){
            for (dev = 0; dev < deviceCount; dev++){
                if(i+dev<nangles){
                    geoArray[sp].alpha=angles[(i+dev)*3];
                    geoArray[sp].theta=angles[(i+dev)*3+1];
                    geoArray[sp].psi  =angles[(i+dev)*3+2];
                    //precomute distances for faster execution
                    //Precompute per angle constant stuff for speed
                    computeDeltas(geoArray[sp],i+dev, &uvOrigin, &deltaU, &deltaV, &source);
                    
                    // compute upper bound limits of ray tracing
                    cropdist_init=maxdistanceCuboid(geo,i+dev); // TODO: this needs reworking for 3D

                    hipSetDevice(dev);
                   //TODO: we could do this around X and Y axis too, but we would need to compute the new axis of rotation (not possible to know from jsut the angles)
                    if (geo.theta==0.0f & geo.psi==0.0f){
                        kernelPixelDetector<false><<<grid,block>>>(geoArray[sp],dProjection[dev], source, deltaU, deltaV, uvOrigin,geo.DSO[i+dev],floor(cropdist_init),texImg[dev]);
                    }
                    else{
                        kernelPixelDetector<true><<<grid,block>>>(geoArray[sp],dProjection[dev], source, deltaU, deltaV, uvOrigin,geo.DSO[i+dev],floor(cropdist_init),texImg[dev]);
                    }
                            //cudaCheckErrors("Kernel fail");
                }
                
            }
            for (dev = 0; dev < deviceCount; dev++){
                if(i+dev<nangles){
                    hipSetDevice(dev);
                    if (!fits_in_memory){
                        hipMemcpyAsync(dProjection_accum[dev], result[i+dev], num_bytes_proj, hipMemcpyHostToDevice);
                        vecAddInPlaceInterp<<<(geo.nDetecU*geo.nDetecV+MAXTREADS-1)/MAXTREADS,MAXTREADS>>>(dProjection[dev],dProjection_accum[dev],(unsigned long)geo.nDetecU*geo.nDetecV );
                    }
                }
            }
            for (dev = 0; dev < deviceCount; dev++){
                if(i+dev<nangles){
                    // copy result to host
                    hipSetDevice(dev);
                    hipMemcpyAsync(result[i+dev], dProjection[dev], num_bytes_proj, hipMemcpyDeviceToHost);
                }
                
            }
            cudaCheckErrors("hipMemcpy output fail");
        }
        
        for (dev = 0; dev < deviceCount; dev++){
            hipSetDevice(dev);
            hipDestroyTextureObject(texImg[dev]);
            hipFreeArray(d_cuArrTex[dev]);
            
        }
    }
    
    
    for (dev = 0; dev < deviceCount; dev++){
        hipSetDevice(dev);
        hipFree(dProjection[dev]);
        
    }
    free(dProjection);
    
    if(!fits_in_memory){
        for (dev = 0; dev < deviceCount; dev++){
            hipSetDevice(dev);
            hipFree(dProjection_accum[dev]);
            
        }
        free(dProjection_accum);
    }
    cudaCheckErrors("hipFree d_imagedata fail");
    
    hipDeviceReset();
    return 0;
}
void CreateTextureInterp(int num_devices,const float* imagedata,Geometry geo,hipArray** d_cuArrTex, hipTextureObject_t *texImage)
{
    //size_t size_image=geo.nVoxelX*geo.nVoxelY*geo.nVoxelZ;
    for (unsigned int i = 0; i < num_devices; i++){
        hipSetDevice(i);
        
        //hipArray Descriptor
        const hipExtent extent = make_hipExtent(geo.nVoxelX, geo.nVoxelY, geo.nVoxelZ);
        hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
        //cuda Array
        hipMalloc3DArray(&d_cuArrTex[i], &channelDesc, extent);
        cudaCheckErrors("Texture memory allocation fail");
        hipMemcpy3DParms copyParams = {0};
        
        
        //Array creation
        copyParams.srcPtr   = make_hipPitchedPtr((void *)imagedata, extent.width*sizeof(float), extent.width, extent.height);
        copyParams.dstArray = d_cuArrTex[i];
        copyParams.extent   = extent;
        copyParams.kind     = hipMemcpyHostToDevice;
        hipMemcpy3D(&copyParams);
        cudaCheckErrors("Texture memory data copy fail");
        //Array creation End
        
        hipResourceDesc    texRes;
        memset(&texRes, 0, sizeof(hipResourceDesc));
        texRes.resType = hipResourceTypeArray;
        texRes.res.array.array  = d_cuArrTex[i];
        hipTextureDesc     texDescr;
        memset(&texDescr, 0, sizeof(hipTextureDesc));
        texDescr.normalizedCoords = false;
        if (geo.accuracy>1){
            texDescr.filterMode = hipFilterModePoint;
            geo.accuracy=1;
        }
        else{
            texDescr.filterMode = hipFilterModeLinear;
        }
        texDescr.addressMode[0] = hipAddressModeBorder;
        texDescr.addressMode[1] = hipAddressModeBorder;
        texDescr.addressMode[2] = hipAddressModeBorder;
        texDescr.readMode = hipReadModeElementType;
        hipCreateTextureObject(&texImage[i], &texRes, &texDescr, NULL);
        cudaCheckErrors("Texture object creation fail");
    }
}

/* This code generates the geometries needed to split the image properly in
 * cases where the entire image does not fit in the memory of the GPU
 **/
void splitImageInterp(unsigned int splits,Geometry geo,Geometry* geoArray, unsigned int nangles){
    
    unsigned long splitsize=(geo.nVoxelZ+splits-1)/splits;// ceil if not divisible
    for(unsigned int sp=0;sp<splits;sp++){
        geoArray[sp]=geo;
        // All of them are splitsize, but the last one, possible
        geoArray[sp].nVoxelZ=((sp+1)*splitsize<geo.nVoxelZ)?  splitsize:  geo.nVoxelZ-splitsize*sp;
        geoArray[sp].sVoxelZ= geoArray[sp].nVoxelZ* geoArray[sp].dVoxelZ;
        
        // We need to redefine the offsets, as now each subimage is not aligned in the origin.
        geoArray[sp].offOrigZ=(float *)malloc(nangles*sizeof(float));
        for (unsigned int i=0;i<nangles;i++){
            geoArray[sp].offOrigZ[i]=geo.offOrigZ[i]-geo.sVoxelZ/2+sp*geoArray[0].sVoxelZ+geoArray[sp].sVoxelZ/2;
        }
        
    }
}



/* This code precomputes The location of the source and the Delta U and delta V (in the warped space)
 * to compute the locations of the x-rays. While it seems verbose and overly-optimized,
 * it does saves about 30% of each of the kernel calls. Thats something!
 **/
void computeDeltas(Geometry geo,unsigned int i, Point3D* uvorigin, Point3D* deltaU, Point3D* deltaV, Point3D* source){
    Point3D S;
    S.x=geo.DSO[i];
    S.y=0;
    S.z=0;
    
    //End point
    Point3D P,Pu0,Pv0;
    
    P.x  =-(geo.DSD[i]-geo.DSO[i]);   P.y  = geo.dDetecU*(0-((float)geo.nDetecU/2)+0.5);       P.z  = geo.dDetecV*(((float)geo.nDetecV/2)-0.5-0);
    Pu0.x=-(geo.DSD[i]-geo.DSO[i]);   Pu0.y= geo.dDetecU*(1-((float)geo.nDetecU/2)+0.5);       Pu0.z= geo.dDetecV*(((float)geo.nDetecV/2)-0.5-0);
    Pv0.x=-(geo.DSD[i]-geo.DSO[i]);   Pv0.y= geo.dDetecU*(0-((float)geo.nDetecU/2)+0.5);       Pv0.z= geo.dDetecV*(((float)geo.nDetecV/2)-0.5-1);
    // Geomtric trasnformations:
    
    
    // Now we have the Real world (OXYZ) coordinates of the bottom corner and its two neighbours.
    // The obkjective is to get a position of the detector in a coordinate system where:
    // 1-units are voxel size (in each direction can be different)
    // 2-The image has the its first voxel at (0,0,0)
    // 3-The image never rotates
    
    // To do that, we need to compute the "deltas" the detector, or "by how much
    // (in new xyz) does the voxels change when and index is added". To do that
    // several geometric steps needs to be changed
    
    //1.Roll,pitch,jaw
    // The detector can have a small rotation.
    // according to
    //"A geometric calibration method for cone beam CT systems" Yang K1, Kwan AL, Miller DF, Boone JM. Med Phys. 2006 Jun;33(6):1695-706.
    // Only the Z rotation will have a big influence in the image quality when they are small.
    // Still all rotations are supported
    
    // To roll pitch jaw, the detector has to be in centered in OXYZ.
    P.x=0;Pu0.x=0;Pv0.x=0;
    
    // Roll pitch yaw
    rollPitchYaw(geo,i,&P);
    rollPitchYaw(geo,i,&Pu0);
    rollPitchYaw(geo,i,&Pv0);
    //Now ltes translate the detector coordinates to DOD (original position on real coordinate system:
    P.x=P.x-(geo.DSD[i]-geo.DSO[i]);
    Pu0.x=Pu0.x-(geo.DSD[i]-geo.DSO[i]);
    Pv0.x=Pv0.x-(geo.DSD[i]-geo.DSO[i]);
    //2: Offset detector
    
    
    //S doesnt need to chagne
    
    
    
    //3: Rotate around RZ RY RZ
    Point3D Pfinal, Pfinalu0, Pfinalv0;
    Pfinal.x  =P.x;
    Pfinal.y  =P.y  +geo.offDetecU[i]; Pfinal.z  =P.z  +geo.offDetecV[i];
    Pfinalu0.x=Pu0.x;
    Pfinalu0.y=Pu0.y  +geo.offDetecU[i]; Pfinalu0.z  =Pu0.z  +geo.offDetecV[i];
    Pfinalv0.x=Pv0.x;
    Pfinalv0.y=Pv0.y  +geo.offDetecU[i]; Pfinalv0.z  =Pv0.z  +geo.offDetecV[i];
    
    eulerZYZ(geo,&Pfinal);
    eulerZYZ(geo,&Pfinalu0);
    eulerZYZ(geo,&Pfinalv0);
    eulerZYZ(geo,&S);
    
    
    //3: Offset image (instead of offseting image, -offset everything else)
    
    Pfinal.x  =Pfinal.x-geo.offOrigX[i];     Pfinal.y  =Pfinal.y-geo.offOrigY[i];     Pfinal.z  =Pfinal.z-geo.offOrigZ[i];
    Pfinalu0.x=Pfinalu0.x-geo.offOrigX[i];   Pfinalu0.y=Pfinalu0.y-geo.offOrigY[i];   Pfinalu0.z=Pfinalu0.z-geo.offOrigZ[i];
    Pfinalv0.x=Pfinalv0.x-geo.offOrigX[i];   Pfinalv0.y=Pfinalv0.y-geo.offOrigY[i];   Pfinalv0.z=Pfinalv0.z-geo.offOrigZ[i];
    S.x=S.x-geo.offOrigX[i];                 S.y=S.y-geo.offOrigY[i];                 S.z=S.z-geo.offOrigZ[i];
    
    // As we want the (0,0,0) to be in a corner of the image, we need to translate everything (after rotation);
    Pfinal.x  =Pfinal.x+geo.sVoxelX/2-geo.dVoxelX/2;      Pfinal.y  =Pfinal.y+geo.sVoxelY/2-geo.dVoxelY/2;          Pfinal.z  =Pfinal.z  +geo.sVoxelZ/2-geo.dVoxelZ/2;
    Pfinalu0.x=Pfinalu0.x+geo.sVoxelX/2-geo.dVoxelX/2;    Pfinalu0.y=Pfinalu0.y+geo.sVoxelY/2-geo.dVoxelY/2;        Pfinalu0.z=Pfinalu0.z+geo.sVoxelZ/2-geo.dVoxelZ/2;
    Pfinalv0.x=Pfinalv0.x+geo.sVoxelX/2-geo.dVoxelX/2;    Pfinalv0.y=Pfinalv0.y+geo.sVoxelY/2-geo.dVoxelY/2;        Pfinalv0.z=Pfinalv0.z+geo.sVoxelZ/2-geo.dVoxelZ/2;
    S.x       =S.x+geo.sVoxelX/2-geo.dVoxelX/2;           S.y       =S.y+geo.sVoxelY/2-geo.dVoxelY/2;               S.z       =S.z      +geo.sVoxelZ/2-geo.dVoxelZ/2;
    
    //4. Scale everything so dVoxel==1
    Pfinal.x  =Pfinal.x/geo.dVoxelX;      Pfinal.y  =Pfinal.y/geo.dVoxelY;        Pfinal.z  =Pfinal.z/geo.dVoxelZ;
    Pfinalu0.x=Pfinalu0.x/geo.dVoxelX;    Pfinalu0.y=Pfinalu0.y/geo.dVoxelY;      Pfinalu0.z=Pfinalu0.z/geo.dVoxelZ;
    Pfinalv0.x=Pfinalv0.x/geo.dVoxelX;    Pfinalv0.y=Pfinalv0.y/geo.dVoxelY;      Pfinalv0.z=Pfinalv0.z/geo.dVoxelZ;
    S.x       =S.x/geo.dVoxelX;           S.y       =S.y/geo.dVoxelY;             S.z       =S.z/geo.dVoxelZ;
    
    
    //mexPrintf("COR: %f \n",geo.COR[i]);
    //5. apply COR. Wherever everything was, now its offesetd by a bit.
//     Only wors for standard rotaiton, not aribtary axis rotation.
    float CORx, CORy;
    CORx=-geo.COR[i]*sin(geo.alpha)/geo.dVoxelX;
    CORy= geo.COR[i]*cos(geo.alpha)/geo.dVoxelY;
    Pfinal.x+=CORx;   Pfinal.y+=CORy;
    Pfinalu0.x+=CORx;   Pfinalu0.y+=CORy;
    Pfinalv0.x+=CORx;   Pfinalv0.y+=CORy;
    S.x+=CORx; S.y+=CORy;
    
    // return
    
    *uvorigin=Pfinal;
    
    deltaU->x=Pfinalu0.x-Pfinal.x;
    deltaU->y=Pfinalu0.y-Pfinal.y;
    deltaU->z=Pfinalu0.z-Pfinal.z;
    
    deltaV->x=Pfinalv0.x-Pfinal.x;
    deltaV->y=Pfinalv0.y-Pfinal.y;
    deltaV->z=Pfinalv0.z-Pfinal.z;
    
    *source=S;
}

float maxdistanceCuboid(Geometry geo,unsigned int i){
    ///////////
    // Compute initial "t" so we access safely as less as out of bounds as possible.
    //////////
    
    
    float maxCubX,maxCubY,maxCubZ;
    // Forgetting Z, compute mas distance: diagonal+offset
    maxCubX=(geo.nVoxelX/2+ abs(geo.offOrigX[i])/geo.dVoxelX);
    maxCubY=(geo.nVoxelY/2+ abs(geo.offOrigY[i])/geo.dVoxelY);
    maxCubZ=(geo.nVoxelZ/2+ abs(geo.offOrigZ[i])/geo.dVoxelZ);
    
    float a,b;
    a=geo.DSO[i]/geo.dVoxelX;
    b=geo.DSO[i]/geo.dVoxelY;
    
//  As the return of this value is in "voxel space", the source may have an elliptical curve.
//  The distance returned is the safe distance that can be skipped for a given angle alpha, before we need to start sampling.
    
    if (geo.theta==0.0f & geo.psi==0.0f) // Special case, it will make the code faster
        return max(a*b/sqrt(a*a*sin(geo.alpha)*sin(geo.alpha)+b*b*cos(geo.alpha)*cos(geo.alpha))-
                sqrt(maxCubX*maxCubX+maxCubY*maxCubY),0.0f);
    //TODO: think of more special cases?
    return max(geo.DSO[i]/max(max(geo.dVoxelX,geo.dVoxelY),geo.dVoxelZ)-sqrt(maxCubX*maxCubX+maxCubY*maxCubY+maxCubZ*maxCubZ),0.0f);

}
void rollPitchYaw(Geometry geo,unsigned int i, Point3D* point){
    Point3D auxPoint;
    auxPoint.x=point->x;
    auxPoint.y=point->y;
    auxPoint.z=point->z;
    
    point->x=cos(geo.dRoll[i])*cos(geo.dPitch[i])*auxPoint.x
            +(cos(geo.dRoll[i])*sin(geo.dPitch[i])*sin(geo.dYaw[i]) - sin(geo.dRoll[i])*cos(geo.dYaw[i]))*auxPoint.y
            +(cos(geo.dRoll[i])*sin(geo.dPitch[i])*cos(geo.dYaw[i]) + sin(geo.dRoll[i])*sin(geo.dYaw[i]))*auxPoint.z;
    
    point->y=sin(geo.dRoll[i])*cos(geo.dPitch[i])*auxPoint.x
            +(sin(geo.dRoll[i])*sin(geo.dPitch[i])*sin(geo.dYaw[i]) + cos(geo.dRoll[i])*cos(geo.dYaw[i]))*auxPoint.y
            +(sin(geo.dRoll[i])*sin(geo.dPitch[i])*cos(geo.dYaw[i]) - cos(geo.dRoll[i])*sin(geo.dYaw[i]))*auxPoint.z;
    
    point->z=-sin(geo.dPitch[i])*auxPoint.x
            +cos(geo.dPitch[i])*sin(geo.dYaw[i])*auxPoint.y
            +cos(geo.dPitch[i])*cos(geo.dYaw[i])*auxPoint.z;
    
}
void eulerZYZ(Geometry geo,  Point3D* point){
    Point3D auxPoint;
    auxPoint.x=point->x;
    auxPoint.y=point->y;
    auxPoint.z=point->z;
    
    point->x=(+cos(geo.alpha)*cos(geo.theta)*cos(geo.psi)-sin(geo.alpha)*sin(geo.psi))*auxPoint.x+
            (-cos(geo.alpha)*cos(geo.theta)*sin(geo.psi)-sin(geo.alpha)*cos(geo.psi))*auxPoint.y+
            cos(geo.alpha)*sin(geo.theta)*auxPoint.z;
    
    point->y=(+sin(geo.alpha)*cos(geo.theta)*cos(geo.psi)+cos(geo.alpha)*sin(geo.psi))*auxPoint.x+
            (-sin(geo.alpha)*cos(geo.theta)*sin(geo.psi)+cos(geo.alpha)*cos(geo.psi))*auxPoint.y+
            sin(geo.alpha)*sin(geo.theta)*auxPoint.z;
    
    point->z=-sin(geo.theta)*cos(geo.psi)*auxPoint.x+
            sin(geo.theta)*sin(geo.psi)*auxPoint.y+
            cos(geo.theta)*auxPoint.z;
    
    
}
