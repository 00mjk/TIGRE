#include "hip/hip_runtime.h"
/*
 * Code that uses texture memory to compute a 3D projection of CBCT 
 *
 * IMPORTANT!!! CAUTION!! This code is designed for a Tesla 40k GPU.
 * It is a safe assumption to say that this code wont work in other GPUs as expected
 * or at all. Some of the involved reasons: float/double arithmetic.
 *
 * Ander Biguri
 */

#include <algorithm>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "projection.hpp"
#include "mex.h"
#include <math.h>

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            mexPrintf("%s \n",msg);\
            mexErrMsgIdAndTxt("CBCT:CUDA:projection",hipGetErrorString(__err));\
		        } \
	    } while (0)
            

// Declare the texture reference.
texture<float, hipTextureType3D , hipReadModeElementType> tex; 

#define MAXTREADS 1024
/*GEOMETRY DEFINITION
 *               
 *                Detector plane, behind
 *            |-----------------------------| 
 *            |                             | 
 *            |                             | 
 *            |                             | 
 *            |                             | 
 *            |      +--------+             |
              |     /        /|             |
     A Z      |    /        / |*D           |
     |        |   +--------+  |             |
     |        |   |        |  |             |
     |        |   |     *O |  +             |
     *--->y   |   |        | /              |
    /         |   |        |/               |
   V X        |   +--------+                |
 *            |-----------------------------|
 *   
 *           *S
 *
 *
 *
 *
 *
 **/


__global__ void kernelPixelDetector( Geometry geo,
                                    double* detector,
                                    Point3D source ,
                                    Point3D deltaU, 
                                    Point3D deltaV,
                                    Point3D uvOrigin){
   
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx>= geo.nDetecU* geo.nDetecV)
        return;
    
    
   

    /////////////////////////////// Get pixel coords
   int pixelV = geo.nDetecV-idx % geo.nDetecV-1;
   int pixelU = idx / geo.nDetecV;
    
    

    double vectX,vectY,vectZ;
    Point3D P;
    P.x=(uvOrigin.x+pixelU*deltaU.x+pixelV*deltaV.x);
    P.y=(uvOrigin.y+pixelU*deltaU.y+pixelV*deltaV.y);
    P.z=(uvOrigin.z+pixelU*deltaU.z+pixelV*deltaV.z);
    
    
    double length=sqrt((source.x-P.x)*(source.x-P.x)+(source.y-P.y)*(source.y-P.y)+(source.z-P.z)*(source.z-P.z));
    length=ceil(length/geo.accuracy);//Divide the directional vector by an integer
    vectX=(P.x -source.x)/(length); 
    vectY=(P.y -source.y)/(length); 
    vectZ=(P.z -source.z)/(length);
    
//     //here comes the deal
    double x,y,z;
    double sum=0;
    double i;
    
    double deltalength=sqrt((vectX*geo.dVoxelX)*(vectX*geo.dVoxelX)+
                            (vectY*geo.dVoxelY)*(vectY*geo.dVoxelY)+
                            (vectZ*geo.dVoxelZ)*(vectZ*geo.dVoxelZ) );

    for (i=0; i<=length; i=i+1){
        x=vectX*(double)i+source.x;
        y=vectY*(double)i+source.y;
        z=vectZ*(double)i+source.z;

        sum += (double)tex3D(tex, x+0.5, y+0.5, z+0.5);

    }
    detector[idx]=sum*deltalength;
}



int projection(float const * const img, Geometry geo, double** result,double const * const alphas,int nalpha){

   
    // BEFORE DOING ANYTHING: Use the proper CUDA enabled GPU: Tesla K40c
    
    // If you have another GPU and want to use this code, please change it, but make sure you know that is compatible.
    // also change MAXTREADS
    
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
     if (deviceCount == 0)
    {
        mexErrMsgIdAndTxt("CBCT:CUDA:Ax:hipGetDeviceCount","No CUDA enabled NVIDIA GPUs found");
    }
    bool found=false;
    for (int dev = 0; dev < deviceCount; ++dev)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        if (strcmp(deviceProp.name, "Tesla K40c") == 0){
            hipSetDevice(dev);
            found=true;
            break;
        }
    }
    if (!found)
        mexErrMsgIdAndTxt("CBCT:CUDA:Ax:cudaDevice","No Tesla K40c found");
    // DONE, Tesla found

    // copy data to CUDA memory
    hipArray *d_imagedata = 0;

    const hipExtent extent = make_hipExtent(geo.nVoxelX, geo.nVoxelY, geo.nVoxelZ);
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	hipMalloc3DArray(&d_imagedata, &channelDesc, extent);
	cudaCheckErrors("hipMalloc3D error 3D tex");
    
    hipMemcpy3DParms copyParams = { 0 };
	copyParams.srcPtr = make_hipPitchedPtr((void*)img, extent.width*sizeof(float), extent.width, extent.height);
	copyParams.dstArray = d_imagedata;
	copyParams.extent = extent;
	copyParams.kind = hipMemcpyHostToDevice;
	hipMemcpy3D(&copyParams);
    
	cudaCheckErrors("hipMemcpy3D fail");
    
    // Configure texture options
    tex.normalized = false;
	tex.filterMode = hipFilterModeLinear;
	tex.addressMode[0] = hipAddressModeBorder;
	tex.addressMode[1] = hipAddressModeBorder;
	tex.addressMode[2] = hipAddressModeBorder;
    
    hipBindTextureToArray(tex, d_imagedata, channelDesc);    
       
	cudaCheckErrors("3D texture memory bind fail"); 
    

    //Done! Image put into texture memory.
    

    size_t num_bytes = geo.nDetecU*geo.nDetecV * sizeof(double);
    double* dProjection;
    hipMalloc((void**)&dProjection, num_bytes);
    cudaCheckErrors("hipMalloc fail");
    

    Point3D source, deltaU, deltaV, uvOrigin;
  
    for (int i=0;i<nalpha;i++){
        
        geo.alpha=alphas[i];
        //Precompute per angle constant stuff for speed
        computeDeltas(geo,geo.alpha,i, &uvOrigin, &deltaU, &deltaV, &source);
        //Ray tracing!  
        kernelPixelDetector<<<(geo.nDetecU*geo.nDetecV + MAXTREADS-1) / MAXTREADS,MAXTREADS>>>(geo,dProjection, source, deltaU, deltaV, uvOrigin);
      

        cudaCheckErrors("Kernel fail");
         // copy result to host
        hipMemcpy(result[i], dProjection, num_bytes, hipMemcpyDeviceToHost);
        cudaCheckErrors("hipMemcpy fail");
        

    }

     hipUnbindTexture(tex);
     cudaCheckErrors("Unbind  fail");
     
     hipFree(dProjection);
     hipFreeArray(d_imagedata);
     cudaCheckErrors("hipFree d_imagedata fail");


     
     
     
 return 0;   
}




/* This code precomputes The location of the source and the Delta U and delta V (in the warped space) 
 * to compute the locations of the x-rays. While it seems verbose and overly-optimized, 
 * it does saves about 30% of each of the kernel calls. Thats something!
 **/
void computeDeltas(Geometry geo, double alpha,int i, Point3D* uvorigin, Point3D* deltaU, Point3D* deltaV, Point3D* source){
    Point3D S;
    S.x=geo.DSO;
    S.y=0;
    S.z=0;
    
    //End point
    Point3D P,Pu0,Pv0;
    
    P.x  =-(geo.DSD-geo.DSO);   P.y  = geo.dDetecU*(0-(double)(geo.nDetecU/2)+0.5);       P.z  = geo.dDetecV*((double)(geo.nDetecV/2)-0.5-0);
    Pu0.x=-(geo.DSD-geo.DSO);   Pu0.y= geo.dDetecU*(1-(double)(geo.nDetecU/2)+0.5);       Pu0.z= geo.dDetecV*((double)(geo.nDetecV/2)-0.5-0);  
    Pv0.x=-(geo.DSD-geo.DSO);   Pv0.y= geo.dDetecU*(0-(double)(geo.nDetecU/2)+0.5);       Pv0.z= geo.dDetecV*((double)(geo.nDetecV/2)-0.5-1);
    // Geomtric trasnformations:
    
    //1: Offset detector
       
    //P.x
    P.y  =P.y  +geo.offDetecU[i];    P.z  =P.z  +geo.offDetecV[i];
    Pu0.y=Pu0.y+geo.offDetecU[i];    Pu0.z=Pu0.z+geo.offDetecV[i];
    Pv0.y=Pv0.y+geo.offDetecU[i];    Pv0.z=Pv0.z+geo.offDetecV[i];
    //S doesnt need to chagne
    
    
    //3: Rotate (around z)!
    Point3D Pfinal, Pfinalu0, Pfinalv0;  
    
    Pfinal.x  =P.x*cos(geo.alpha)-P.y*sin(geo.alpha);       Pfinal.y  =P.y*cos(geo.alpha)+P.x*sin(geo.alpha);       Pfinal.z  =P.z;
    Pfinalu0.x=Pu0.x*cos(geo.alpha)-Pu0.y*sin(geo.alpha);   Pfinalu0.y=Pu0.y*cos(geo.alpha)+Pu0.x*sin(geo.alpha);   Pfinalu0.z=Pu0.z;
    Pfinalv0.x=Pv0.x*cos(geo.alpha)-Pv0.y*sin(geo.alpha);   Pfinalv0.y=Pv0.y*cos(geo.alpha)+Pv0.x*sin(geo.alpha);   Pfinalv0.z=Pv0.z;
    
    Point3D S2; 
    S2.x=S.x*cos(geo.alpha)-S.y*sin(geo.alpha);
    S2.y=S.y*cos(geo.alpha)+S.x*sin(geo.alpha);
    S2.z=S.z;
    
    //2: Offset image (instead of offseting image, -offset everything else)
    
    Pfinal.x  =Pfinal.x-geo.offOrigX[i];     Pfinal.y  =Pfinal.y-geo.offOrigY[i];     Pfinal.z  =Pfinal.z-geo.offOrigZ[i];
    Pfinalu0.x=Pfinalu0.x-geo.offOrigX[i];   Pfinalu0.y=Pfinalu0.y-geo.offOrigY[i];   Pfinalu0.z=Pfinalu0.z-geo.offOrigZ[i];
    Pfinalv0.x=Pfinalv0.x-geo.offOrigX[i];   Pfinalv0.y=Pfinalv0.y-geo.offOrigY[i];   Pfinalv0.z=Pfinalv0.z-geo.offOrigZ[i];   
    S2.x=S2.x-geo.offOrigX[i];       S2.y=S2.y-geo.offOrigY[i];       S2.z=S2.z-geo.offOrigZ[i];
    
    // As we want the (0,0,0) to be in a corner of the image, we need to translate everything (after rotation);
    Pfinal.x  =Pfinal.x+geo.sVoxelX/2-geo.dVoxelX/2;      Pfinal.y  =Pfinal.y+geo.sVoxelY/2-geo.dVoxelY/2;          Pfinal.z  =Pfinal.z  +geo.sVoxelZ/2-geo.dVoxelZ/2;
    Pfinalu0.x=Pfinalu0.x+geo.sVoxelX/2-geo.dVoxelX/2;    Pfinalu0.y=Pfinalu0.y+geo.sVoxelY/2-geo.dVoxelY/2;        Pfinalu0.z=Pfinalu0.z+geo.sVoxelZ/2-geo.dVoxelZ/2;
    Pfinalv0.x=Pfinalv0.x+geo.sVoxelX/2-geo.dVoxelX/2;    Pfinalv0.y=Pfinalv0.y+geo.sVoxelY/2-geo.dVoxelY/2;        Pfinalv0.z=Pfinalv0.z+geo.sVoxelZ/2-geo.dVoxelZ/2;
    S2.x      =S2.x+geo.sVoxelX/2-geo.dVoxelX/2;          S2.y      =S2.y+geo.sVoxelY/2-geo.dVoxelY/2;              S2.z      =S2.z      +geo.sVoxelZ/2-geo.dVoxelZ/2;
    
    //4. Scale everything so dVoxel==1
    Pfinal.x  =Pfinal.x/geo.dVoxelX;      Pfinal.y  =Pfinal.y/geo.dVoxelY;        Pfinal.z  =Pfinal.z/geo.dVoxelZ;
    Pfinalu0.x=Pfinalu0.x/geo.dVoxelX;    Pfinalu0.y=Pfinalu0.y/geo.dVoxelY;      Pfinalu0.z=Pfinalu0.z/geo.dVoxelZ;
    Pfinalv0.x=Pfinalv0.x/geo.dVoxelX;    Pfinalv0.y=Pfinalv0.y/geo.dVoxelY;      Pfinalv0.z=Pfinalv0.z/geo.dVoxelZ;
    S2.x      =S2.x/geo.dVoxelX;          S2.y      =S2.y/geo.dVoxelY;            S2.z      =S2.z/geo.dVoxelZ;   
    
    // return
    
    *uvorigin=Pfinal;
    
    deltaU->x=Pfinalu0.x-Pfinal.x;
    deltaU->y=Pfinalu0.y-Pfinal.y;
    deltaU->z=Pfinalu0.z-Pfinal.z;
    
    deltaV->x=Pfinalv0.x-Pfinal.x;
    deltaV->y=Pfinalv0.y-Pfinal.y;
    deltaV->z=Pfinalv0.z-Pfinal.z;
    
    *source=S2;
}





/////////////////////
///////////////////// The code below is not used.
/////////////////////
/////////////////////
/////////////////////
/////////////////////
/////////////////////
/////////////////////
/////////////////////
/////////////////////
// double computeMaxLength(Geometry geo, double alpha){ // Ander: I like alpha as an argument tomake sure the programer puts it in. Explicit call. 
//     
//     //Start point
//     Point3D S;
//     S.x=geo.DSO;
//     S.y=0;
//     S.z=0;
//     
//     //End point
//     Point3D P;
//     P.x=-(geo.DSD-geo.DSO);
//     P.y= geo.dDetecU*(0-(double)(geo.nDetecU/2)+0.5);
//     P.z= geo.dDetecV*((double)(geo.nDetecV/2)+0.5-0);
//     
//     // Geomtric trasnformations:
//     
//     //1: Offset detector
//        
//     //P.x
//     P.y=P.y+geo.offDetecU;
//     P.z=P.z+geo.offDetecV;
//     //S doesnt need to chagne
//     
//     //2: Offset image (instead of offseting image, -offset everything else)
//     
//     P.x=P.x-geo.offOrigX;
//     P.y=P.y-geo.offOrigY;
//     P.z=P.z-geo.offOrigZ;
//     
//     S.x=S.x-geo.offOrigX;
//     S.y=S.y-geo.offOrigY;
//     S.z=S.z-geo.offOrigZ;
//     
//     //3: Rotate (around z)!
//     Point3D P2;   
//     P2.x=P.x*cos(alpha)-P.y*sin(alpha);
//     P2.y=P.y*cos(alpha)+P.x*sin(alpha);
//     P2.z=P.z;
//     Point3D S2; 
//     S2.x=S.x*cos(alpha)-S.y*sin(alpha);
//     S2.y=S.y*cos(alpha)+S.x*sin(alpha);
//     S2.z=S.z;
//     // As we want the (0,0,0) to be in a corner of the image, we need to translate everything (after rotation);
//     P2.x=P2.x+geo.sVoxelX/2;
//     P2.y=P2.y+geo.sVoxelY/2;
//     P2.z=P2.z+geo.sVoxelZ/2;
//     
//     S2.x=S2.x+geo.sVoxelX/2;
//     S2.y=S2.y+geo.sVoxelY/2;
//     S2.z=S2.z+geo.sVoxelZ/2;
//     
//     //4. Scale everything so dVoxel==1
//     P2.x=P2.x/geo.dVoxelX;
//     P2.y=P2.y/geo.dVoxelY;
//     P2.z=P2.z/geo.dVoxelZ;
//     S2.x=S2.x/geo.dVoxelX;
//     S2.y=S2.y/geo.dVoxelY;
//     S2.z=S2.z/geo.dVoxelZ;
//     
//     
//     return sqrt((P2.x-S2.x)*(P2.x-S2.x)   +    (P2.y-S2.y)*(P2.y-S2.y) +(P2.z-S2.z)*(P2.z-S2.z) );
// }
// // This function scales the geometrical data so all the image voxels are 1x1x1
// Geometry nomralizeGeometryImage(Geometry geo){
//     
//     Geometry nGeo; //Normalized geometry
//     //Copy input values
//     nGeo=geo;
//     
//     // This is why we are doing this stuff
//     nGeo.dVoxelX=1;
//     nGeo.dVoxelY=1;
//     nGeo.dVoxelZ=1;
//     // Change total size
//     nGeo.sVoxelX=geo.sVoxelX/geo.dVoxelX; //This shoudl be == geo.nVoxelX;
//     nGeo.sVoxelY=geo.sVoxelY/geo.dVoxelY; //This shoudl be == geo.nVoxelY;
//     nGeo.sVoxelZ=geo.sVoxelZ/geo.dVoxelZ; //This shoudl be == geo.nVoxelZ;
//     
//     // As in the beggining U is alinged with Y and V with Z, they also need to be modified.
//     
//     nGeo.dDetecU=geo.dDetecU/geo.dVoxelY;
//     nGeo.dDetecV=geo.dDetecV/geo.dVoxelZ;
// 
//     //Modify DSO and DSD w.r.t. X
//     
//     nGeo.DSO=geo.DSO/geo.dVoxelX;
//     nGeo.DSD=geo.DSD/geo.dVoxelX;
//     
//     // The new "units" have this real size
//     nGeo.unitX=geo.dVoxelX;
//     nGeo.unitY=geo.dVoxelY;
//     nGeo.unitZ=geo.dVoxelZ;
// 
//     //Compute maxlength
//     nGeo.maxLength=sqrt(nGeo.DSD*nGeo.DSD+sqrt(nGeo.sDetecU/2*nGeo.sDetecU/2+nGeo.sDetecV/2*nGeo.sDetecV/2));
// 
//     return nGeo;
//     
// }